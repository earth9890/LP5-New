#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(int *A, int *B, int *C, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size)
    {
        C[tid] = A[tid] + B[tid];
    }
}

void userInput(int *vector, int size)
{
    cout << "Enter " << size << " elements for the vector:\n";
    for (int i = 0; i < size; i++)
    {
        cout << "Element " << i + 1 << ": ";
        cin >> vector[i];
    }
}

void print(int *vector, int size)
{
    for (int i = 0; i < size; i++)
    {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main()
{
    int N;
    int *A, *B, *C;
    cout << "Enter the size of the vector" << endl;
    cin >> N;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    userInput(A, vectorSize);
    userInput(B, vectorSize);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int *X, *Y, *Z;

    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    // Sequential execution
    double sequential_start = clock();
    for (int i = 0; i < N; i++)
    {
        C[i] = A[i] + B[i];
    }
    double sequential_end = clock();
    // Print vector sequential
    cout << "\nAddition of vectors when added sequentially - ";
    print(C, N);
    // Print execution times
    double seq_time = (sequential_end - sequential_start) / CLOCKS_PER_SEC * 1000.0;
    cout << "Sequential Time is - " << seq_time << "ms\n\n";

    // Parallel execution
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    double parallel_start = clock();
    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);
    hipDeviceSynchronize();
    double parallel_end = clock();

    // Copy data back from device to host (optional for verification)
    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);
    // Print vector parallel
    cout << "\nAddition of vectors when added parallely - ";
    print(C, N);
    // Print execution times
    double par_time = (parallel_end - parallel_start) / CLOCKS_PER_SEC * 1000.0;
    cout << "Parallel Time is - " << par_time << "ms";

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}